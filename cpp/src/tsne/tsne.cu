#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "../../src_prims/utils.h"
#include "distances.h"
#include "exact_kernels.h"
#include "tsne/tsne.h"
#include "unary_op.h"
#include "utils.h"

#include "barnes_hut.h"
#include "exact_tsne.h"
#include "spectral/spectral.h"

#define CHECK(x)                                                       \
  ASSERT(x == 0, "cuSolver or cuBLAS failed at line = %d file = %s\n", \
         __LINE__, __FILE__);

#define MIN(a, b) (a > b) ? b : a

namespace ML {

/**
 * @brief Dimensionality reduction via TSNE using either Barnes Hut O(NlogN) or brute force O(N^2).
 * @input param handle: The GPU handle.
 * @input param X: The dataset you want to apply TSNE on.
 * @output param embedding: The final embedding. Will overwrite this internally.
 * @input param n: Number of rows in data X.
 * @input param p: Number of columns in data X.
 * @input param dim: Number of output dimensions for embeddings embedding.
 * @input param n_neighbors: Number of nearest neighbors used.
 * @input param theta: Float between 0 and 1. Tradeoff for speed (0) vs accuracy (1) for Barnes Hut only.
 * @input param epssq: A tiny jitter to promote numerical stability.
 * @input param perplexity: How many nearest neighbors are used during the construction of Pij.
 * @input param perplexity_max_iter: Number of iterations used to construct Pij.
 * @input param perplexity_tol: The small tolerance used for Pij to ensure numerical stability.
 * @input param early_exaggeration: How much early pressure you want the clusters in TSNE to spread out more.
 * @input param exaggeration_iter: How many iterations you want the early pressure to run for.
 * @input param min_gain: Rounds up small gradient updates.
 * @input param pre_learning_rate: The learning rate during the exaggeration phase.
 * @input param post_learning_rate: The learning rate after the exaggeration phase.
 * @input param max_iter: The maximum number of iterations TSNE should run for.
 * @input param min_grad_norm: The smallest gradient norm TSNE should terminate on.
 * @input param pre_momentum: The momentum used during the exaggeration phase.
 * @input param post_momentum: The momentum used after the exaggeration phase.
 * @input param random_state: Set this to -1 for pure random intializations or >= 0 for reproducible outputs.
 * @input param verbose: Whether to print error messages or not.
 * @input param spectral_intialization: Whether to intialize with spectral embedding. Acts like pseudo PCA.
 * @input param barnes_hut: Whether to use the fast Barnes Hut or use the slower exact version.
 */
void TSNE_fit(const cumlHandle &handle, const float *X, float *embedding,
              const int n, const int p, const int dim, int n_neighbors,
              const float theta, const float epssq, float perplexity,
              const int perplexity_max_iter, const float perplexity_tol,
              const float early_exaggeration, const int exaggeration_iter,
              const float min_gain, const float pre_learning_rate,
              const float post_learning_rate, const int max_iter,
              const float min_grad_norm, const float pre_momentum,
              const float post_momentum, const long long random_state,
              const bool verbose, const bool spectral_intialization,
              bool barnes_hut) {
  ASSERT(n > 0 && p > 0 && dim > 0 && n_neighbors > 0 && X != NULL &&
           embedding != NULL,
         "Wrong input args");

  if (dim > 2 and barnes_hut) {
    barnes_hut = false;
    printf(
      "[Warn]  Barnes Hut only works for dim == 2. Switching to exact "
      "solution.\n");
  }
  if (n_neighbors > n) n_neighbors = n;
  if (n_neighbors > 1023) {
    printf("[Warn]  FAISS only supports maximum n_neighbors = 1023.\n");
    n_neighbors = 1023;
  }
  // Perplexity must be less than number of datapoints
  // "How to Use t-SNE Effectively" https://distill.pub/2016/misread-tsne/
  if (perplexity > n) perplexity = n;

  if (verbose) {
    printf("[Info]  Data size = (%d, %d) with dim = %d perplexity = %f\n", n, p,
           dim, perplexity);
    if (perplexity < 5 or perplexity > 50)
      printf(
        "[Warn]  Perplexity should be within ranges (5, 50). Your results "
        "might be a bit strange...\n");
    if (n_neighbors < perplexity * 3.0f)
      printf(
        "[Warn]  # of Nearest Neighbors should be at least 3 * perplexity. "
        "Your results might be a bit strange...\n");
  }

  auto d_alloc = handle.getDeviceAllocator();
  hipStream_t stream = handle.getStream();

  START_TIMER;
  //---------------------------------------------------
  // Get distances
  if (verbose) printf("[Info] Getting distances.\n");
  float *distances =
    (float *)d_alloc->allocate(sizeof(float) * n * n_neighbors, stream);
  long *indices =
    (long *)d_alloc->allocate(sizeof(long) * n * n_neighbors, stream);
  TSNE::get_distances(X, n, p, indices, distances, n_neighbors, stream);
  //---------------------------------------------------
  END_TIMER(DistancesTime);

  START_TIMER;
  //---------------------------------------------------
  // Normalize distances
  if (verbose)
    printf("[Info] Now normalizing distances so exp(D) doesn't explode.\n");
  TSNE::normalize_distances(n, distances, n_neighbors, stream);
  //---------------------------------------------------
  END_TIMER(NormalizeTime);

  START_TIMER;
  //---------------------------------------------------
  // Optimal perplexity
  if (verbose)
    printf("[Info] Searching for optimal perplexity via bisection search.\n");
  float *P =
    (float *)d_alloc->allocate(sizeof(float) * n * n_neighbors, stream);
  const float P_sum =
    TSNE::perplexity_search(distances, P, perplexity, perplexity_max_iter,
                            perplexity_tol, n, n_neighbors, handle);
  d_alloc->deallocate(distances, sizeof(float) * n * n_neighbors, stream);
  if (verbose) printf("[Info] Perplexity sum = %f\n", P_sum);
  //---------------------------------------------------
  END_TIMER(PerplexityTime);

  START_TIMER;
  //---------------------------------------------------
  // Convert data to COO layout
  MLCommon::Sparse::COO<float> COO_Matrix;
  TSNE::symmetrize_perplexity(P, indices, n, n_neighbors, P_sum,
                              early_exaggeration, &COO_Matrix, stream, handle);
  const int NNZ = COO_Matrix.nnz;
  float *VAL = COO_Matrix.vals;
  const int *COL = COO_Matrix.cols;
  const int *ROW = COO_Matrix.rows;
  //---------------------------------------------------
  END_TIMER(SymmetrizeTime);

  // Intialize via Sparse SVD for COO matrices
  int cols = n;
  int oversamples = 10;
  int k = MIN(2 + oversamples, cols);
  hipsolverHandle_t cusolverH = NULL;
  CHECK(hipsolverDnCreate(&cusolverH));
  hipblasHandle_t cublasH = NULL;
  CHECK(hipblasCreate(&cublasH));

  float *Y /*(n,k)*/ =
    (float *)d_alloc->allocate(sizeof(float) * n * k, stream);
  float *Z /*(p,k)*/ =
    (float *)d_alloc->allocate(sizeof(float) * cols * k, stream);
  random_vector(Z, 0.0f, 1.0f, cols * k, stream, random_state,
                true);  // normal = true

  // Y, _ = np.linalg.qr(Y)
  int lwork_Y = 0;
  CHECK(hipsolverDnSgeqrf_bufferSize(cusolverH, n, k, Y, n, &lwork_Y));
  float *work_Y = (float *)d_alloc->allocate(sizeof(float) * lwork_Y, stream);

  // Z, _ = np.linalg.qr(Z)
  int lwork_Z = 0;
  CHECK(hipsolverDnSgeqrf_bufferSize(cusolverH, cols, k, Z, cols, &lwork_Z));
  float *work_Z = (float *)d_alloc->allocate(sizeof(float) * lwork_Z, stream);

  // Tau for both QR factorizations
  float *tau = (float *)d_alloc->allocate(sizeof(float) * k, stream);
  int *info = (int *)d_alloc->allocate(sizeof(int), stream);

  // Y = X @ Z
  MLCommon::Sparse::coo_gemm(&COO_Matrix, Z, k, Y, stream,
                             false);  // trans = false

  for (int i = 0; i < 3; i++) {
    // Y, _ = np.linalg.qr(Y)
    CHECK(hipsolverDnSgeqrf(cusolverH, n, k, Y, n, tau, work_Y, lwork_Y, info));
    CHECK(
      hipsolverDnSorgqr(cusolverH, n, k, k, Y, n, tau, work_Y, lwork_Y, info));

    // Z = X.T @ Y
    MLCommon::Sparse::coo_gemm(&COO_Matrix, Y, k, Z, stream,
                               true);  // trans = true

    // Z, _ = np.linalg.qr(Z)
    CHECK(hipsolverDnSgeqrf(cusolverH, cols, k, Z, cols, tau, work_Z, lwork_Z,
                           info));
    CHECK(hipsolverDnSorgqr(cusolverH, cols, k, k, Z, cols, tau, work_Z, lwork_Z,
                           info));

    // Y = X @ Z
    MLCommon::Sparse::coo_gemm(&COO_Matrix, Z, k, Y, stream,
                               false);  // trans = false
  }

  // Y, _ = np.linalg.qr(Y)
  CHECK(hipsolverDnSgeqrf(cusolverH, n, k, Y, n, tau, work_Y, lwork_Y, info));
  CHECK(hipsolverDnSorgqr(cusolverH, n, k, k, Y, n, tau, work_Y, lwork_Y, info));

  // Z(p,k) = Y.T @ X (or (X.T @ Y).T)
  MLCommon::Sparse::coo_gemm(&COO_Matrix, Y, k, Z, stream,
                             true);  // trans = true

  // T(k,k) = Z @ Z.T (or (Z.T @ Z))
  float *T = (float *)d_alloc->allocate(sizeof(float) * k * k, stream);

  float alpha = 1.0f;
  float beta = 0.0f;
  hipblasSsyrk(cublasH, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, k, cols, &alpha, Z,
              cols, &beta, T, k);

  // W, Uhat = np.linalg.eigh(T)
  float *W = (float *)d_alloc->allocate(sizeof(float) * k, stream);
  float *Uhat = (float *)d_alloc->allocate(sizeof(float) * k * k, stream);

  int lwork_T = 0;
  CHECK(hipsolverDnSsyevd_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_VECTOR,
                                    HIPBLAS_FILL_MODE_UPPER, k, T, k, W,
                                    &lwork_T));
  float *work_T = (float *)d_alloc->allocate(sizeof(float) * lwork_T, stream);

  CHECK(hipsolverDnSsyevd(cusolverH, HIPSOLVER_EIG_MODE_VECTOR,
                         HIPBLAS_FILL_MODE_UPPER, k, T, k, W, work_T, lwork_T,
                         info));

  int info_cpu;
  CUDA_CHECK(hipMemcpy(&info_cpu, info, sizeof(int), hipMemcpyDeviceToHost));

  printf("Lwork_Y = %d, Lwork_Z = %d lwork_T = %d Info = %d\n", lwork_Y,
         lwork_Z, lwork_T, info_cpu);

  d_alloc->deallocate(work_T, sizeof(float) * lwork_T, stream);
  d_alloc->deallocate(Uhat, sizeof(float) * k * k, stream);
  d_alloc->deallocate(W, sizeof(float) * k, stream);
  d_alloc->deallocate(T, sizeof(float) * k * k, stream);

  d_alloc->deallocate(work_Z, sizeof(float) * lwork_Z, stream);
  d_alloc->deallocate(work_Y, sizeof(float) * lwork_Y, stream);
  d_alloc->deallocate(tau, sizeof(float) * k, stream);
  d_alloc->deallocate(info, sizeof(int), stream);

  d_alloc->deallocate(Y, sizeof(float) * n * k, stream);
  d_alloc->deallocate(Z, sizeof(float) * cols * k, stream);

  hipblasDestroy(cublasH);
  hipsolverDnDestroy(cusolverH);

  if (barnes_hut) {
    TSNE::Barnes_Hut(VAL, COL, ROW, NNZ, handle, embedding, n, theta, epssq,
                     early_exaggeration, exaggeration_iter, min_gain,
                     pre_learning_rate, post_learning_rate, max_iter,
                     min_grad_norm, pre_momentum, post_momentum, random_state,
                     verbose, spectral_intialization);
  } else {
    TSNE::Exact_TSNE(VAL, COL, ROW, NNZ, handle, embedding, n, dim,
                     early_exaggeration, exaggeration_iter, min_gain,
                     pre_learning_rate, post_learning_rate, max_iter,
                     min_grad_norm, pre_momentum, post_momentum, random_state,
                     verbose, spectral_intialization);
  }

  COO_Matrix.destroy();
}

}  // namespace ML
