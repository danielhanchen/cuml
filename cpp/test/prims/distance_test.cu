/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <score/scores.h>
#include <iostream>
#include <vector>
#include "datasets/digits.h"
#include <cuda_utils.h>
#include "distance/distance.h"

#include "common/cuml_allocator.hpp"
#include "common/device_buffer.hpp"

using namespace std;
using namespace MLCommon;
using namespace MLCommon::Score;
using namespace MLCommon::Distance;
using namespace MLCommon::Datasets::Digits;



template <DistanceType d_type, typename T>
static void
get_distances(T *X,
              T *output_D,
              int n,
              int p,
              std::shared_ptr<deviceAllocator> d_alloc,
              hipStream_t stream)
{
  typedef cutlass::Shape<8, 128, 128> OutputTile_t;

  // Determine distance workspace size
  const size_t lwork = getWorkspaceSize<d_type, T, T, T>(X, X, n, n, p);
  void *work = (lwork > 0) ? ((void*) d_alloc->allocate(lwork, stream)) : NULL;
  
  // Find distances
  MLCommon::Distance::distance<d_type, T, T, T, OutputTile_t>(
    X, X, output_D, n, n, p, work, lwork, stream);
  CUDA_CHECK(hipPeekAtLastError());
  
  // Free workspace
  if (lwork > 0) d_alloc->deallocate(work, lwork, stream);
}



template <DistanceType d_type>
class DistanceTest : public ::testing::Test
{
 protected:
  void basicTest()
  {
    hipStream_t stream;
    hipStreamCreate(&stream);

    d_alloc.reset(new defaultDeviceAllocator);

    // Allocate memory
    device_buffer<float> X_d(d_alloc, stream, n*p);
    MLCommon::updateDevice(X_d.data(), digits.data(), n*p, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    device_buffer<float> output_D(d_alloc, stream, n*n);

    // Test each distance type
    get_distances<d_type>(X_d.data(), output_D.data(), n, p, d_alloc, stream);

    hipStreamDestroy(stream);
  }

  void SetUp() override { basicTest(); }

  void TearDown() override {}

 protected:
  int n = 1797;
  int p = 64;
  std::shared_ptr<deviceAllocator> d_alloc;
};


typedef DistanceTest<EucExpandedL2> DistanceTest_EucExpandedL2;
TEST_F(DistanceTest_EucExpandedL2, Result) {}

typedef DistanceTest<EucExpandedL2Sqrt> DistanceTest_EucExpandedL2Sqrt;
TEST_F(DistanceTest_EucExpandedL2Sqrt, Result) {}

typedef DistanceTest<EucExpandedCosine> DistanceTest_EucExpandedCosine;
TEST_F(DistanceTest_EucExpandedCosine, Result) {}

typedef DistanceTest<EucUnexpandedL1> DistanceTest_EucUnexpandedL1;
TEST_F(DistanceTest_EucUnexpandedL1, Result) {}

typedef DistanceTest<EucUnexpandedL2> DistanceTest_EucUnexpandedL2;
TEST_F(DistanceTest_EucUnexpandedL2, Result) {}

typedef DistanceTest<EucUnexpandedL2Sqrt> DistanceTest_EucUnexpandedL2Sqrt;
TEST_F(DistanceTest_EucUnexpandedL2Sqrt, Result) {}
